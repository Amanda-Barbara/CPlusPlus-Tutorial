
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define BLOCK_SIZE 16

void cpu_matrix_multiplication(int* a, int* b, int* c, const int size)
{
    for(int y=0; y<size;++y)
        for(int x=0; x<size; ++x)
        {
            int sum = 0;
            for (size_t i = 0; i < size; i++)
            {
                sum += a[y*size+i]*b[i*size+x];
            }
            c[y*size+x] = sum;
        }
}

__global__ void gpu_matrix_multiplication(int* a, int* b, int* c, const int size)
{
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    int x = blockDim.x*blockIdx.x + threadIdx.x;

    int sum = 0;
    if(y<size && x<size)
    {
        for (size_t i = 0; i < size; i++)
            sum += a[y*size+i]*b[i*size+x];
        c[y*size+x] = sum;
    }
    
}

int main()
{
    int matrix_size = 1000;
    int mem_size = sizeof(int)* matrix_size * matrix_size;
    int *h_a, *h_b, *h_c, *h_cpu;
    hipHostMalloc((void**)&h_a, mem_size, hipHostMallocDefault);
    hipHostMalloc((void**)&h_b, mem_size, hipHostMallocDefault);
    hipHostMalloc((void**)&h_c, mem_size, hipHostMallocDefault);
    hipHostMalloc((void**)&h_cpu, mem_size, hipHostMallocDefault);
    for(int y=0; y<matrix_size;++y)
        for(int x=0;x<matrix_size;++x)
        {
            h_a[y*matrix_size+x] = rand()%1024;
            h_b[y*matrix_size+x] = rand()%1024;
        }
            
    // for(int y=0; y<matrix_size;++y)
    //     for(int x=0;x<matrix_size;++x)
            
    
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, mem_size);
    hipMalloc((void**)&d_b, mem_size);
    hipMalloc((void**)&d_c, mem_size);

    hipMemcpy(d_a, h_a, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, mem_size, hipMemcpyHostToDevice);

    unsigned int grid_rows = (matrix_size+BLOCK_SIZE-1)/BLOCK_SIZE;
    unsigned int grid_cols = (matrix_size+BLOCK_SIZE-1)/BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE); //min warp threads is 32, max is 1024

    gpu_matrix_multiplication<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, matrix_size);

    hipMemcpy(h_c, d_c, mem_size, hipMemcpyDeviceToHost);

    cpu_matrix_multiplication(h_a, h_b, h_cpu, matrix_size);

    

    bool passed = true;

    for(int y=0;y<matrix_size;++y)
        for(int x=0; x<matrix_size;++x)
        {
            if(fabs(h_cpu[y*matrix_size+x]-h_c[y*matrix_size+x])>(1.0e-10))
            passed = false;
        }
    printf("Result:%s\n", passed?"Passed.":"Errors.");
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cpu);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);






    return 0;

}